/* 
* INPUT:
* m: total num of points
* m is in [10, 1000]
* n: n dimensions
* n is in [1,1000]
* k: num of nearest points
* k is in [1,10]
* V: point coordinates
* the integer elements are in [-5,5]
* OUTPUT:
* out: k nearest neighbors
*/

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define INIT_MAX 100000
void showResult(int m, int k, int *out);

extern __shared__ int SMem[];

// compute the square of distance per dimension
// the kth dimension of the ith point and jth point
__device__ void computeDimDist(int i, int j, int n, int *V)
{
	int k = threadIdx.x;
	SMem[k] = (V[i*n+k]-V[j*n+k])*(V[i*n+k]-V[j*n+k]);
}

// compute the square of distance of the ith point and jth point
__global__ void computeDist(int m, int n, int *V, int *D)
{
	int i = blockIdx.x;
   	int j = blockIdx.y;
	int k = threadIdx.x;
	int s;
	int currentScale;
	// calculate the square of distance per dimensions
	// reduce duplicated calculations since d(i, j) = d(j, i)
	// also, we do not consider the trivial case of d(i, i) = 0
	// so we only compute the square distance when i < j 
	if(i < j)
	{
		// use paralel reduction
//		for(s=blockDim.x/2, currentScale=blockDim.x; s>0; s>>=1, currentScale>>=1)
		for(s=blockDim.x/2, currentScale=blockDim.x; s>0; s>>=1)
		computeDimDist(i, j, n, V);
		__syncthreads();
		// use parallel reduction
		for(s=n/2; s>0; s>>=1)
		{
			if(k < s)
			{
				SMem[k] += SMem[k+s];
			}
			__syncthreads();
			// when s is odd, the last element of SMem needs to be added
			if( currentScale>s*2 )
			{
				SMem[0] += SMem[currentScale-1];
			}
			currentScale>>=1;
			__syncthreads();
		}
	}
	D[i*m+j] = SMem[0];
}

__device__ void initSMem(int m, int *D)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	if(i < j)
	{
		SMem[j] = D[i*m+j];
	}
	else
	{
		SMem[j] = D[j*m+i];	
	}
}

__device__ int findMin(int m, int n, int k, int count, int *D, int *out)
{
	__shared__ int R[1000];
	int i = blockIdx.x;
  	int j = threadIdx.x;
	int s = blockDim.x/2;
	int currentScale;
//	int last_s = s;
	int num;
	initSMem(m, D);
	__syncthreads();
	R[j] = j;
	__syncthreads();
	if(j == i)
	{
		SMem[i] = INIT_MAX;
		for(num=0; num<count; num++)
		{
			SMem[ out[i*k+num] ] = INIT_MAX;
		}
	}
	/*
	for(num=0; num<count; num++)
	{
		SMem[ out[i*k+num] ] = INIT_MAX;
	}
	__syncthreads();
	if(j < count)
	{
		SMem[ out[i*k+j] ] = INIT_MAX;
	}
	*/
	__syncthreads();
//	for(s=blockDim.x/2; s>0; s>>=1, last_s=s) 
//	for(s=blockDim.x/2, currentScale=blockDim.x; s>0; s>>=1, currentScale>>=1) 
	for(s=blockDim.x/2, currentScale=blockDim.x; s>0; s>>=1) 
	{
		if(j < s) 
		{
			if(SMem[j] == SMem[j+s])
			{
				if(R[j] > R[j+s])
				{
					R[j] = R[j+s];
				}
			}
			else if(SMem[j] > SMem[j+s])
			{
				SMem[j] = SMem[j+s];
				R[j] = R[j+s];
			}
		}
		__syncthreads();
		if( currentScale>s*2 )
		{
			if(SMem[0] == SMem[currentScale-1])
			{
				if(R[0] > R[currentScale-1])
				{
					R[0] = R[currentScale-1];
				}
			}
			else if(SMem[0] > SMem[currentScale-1])
			{
				SMem[0] = SMem[currentScale-1];
				R[0] = R[currentScale-1];
			}
		}
		currentScale>>=1;
		__syncthreads();
	}
	return R[0];
}

// compute the k nearest neighbors
__global__ void knn(int m, int n, int k, int *V, int *D, int *out)
{
	int i;
	int count;

	i = blockIdx.x;
	__syncthreads();
	for(count=0; count<k; count++)
	{
		out[i*k+count] = findMin(m, n, k, count, D, out);
		__syncthreads();
	}
}

void showResult(int m, int k, int *out)
{
	int i,j;
	for(i=0; i<m; i++)
	{
		for(j=0; j<k; j++)
		{
			printf("%d ", out[i*k+j]);
			if(j == k-1)
			{
				printf("\n");
			}
		}
	} 
} 
int main(int argc, char *argv[]) 
{ 
	int m,n,k;
	int i;
	int *V, *out;				//host copies
	int *d_V, *d_out;			//device copies
	int *D;						
	FILE *fp;
//	FILE *fp_in;
//	FILE *fp_out;
	if(argc != 2)
	{
		printf("Usage: knn <inputfile>\n");
		exit(1);
	}
	if((fp = fopen(argv[1], "r")) == NULL)
	{
		printf("Error open input file!\n");
		exit(1);
	}
	/*
	if((fp_in = fopen(argv[1], "r")) == NULL)
	{
		printf("Error open input file!\n");
		exit(1);
	}
	if((fp_out = fopen("time.txt", "w")) == NULL)
	{
		printf("Error open output file!\n");
		exit(1);
	}
	*/
	while(fscanf(fp, "%d %d %d", &m, &n, &k) != EOF)
	{
		V = (int *) malloc(m*n*sizeof(int));
		out = (int *) malloc(m*k*sizeof(int));

		// allocate space for devices copies
		hipMalloc((void **)&d_V, m*n*sizeof(int));
		hipMalloc((void **)&d_out, m*k*sizeof(int));
		hipMalloc((void **)&D, m*m*sizeof(int));

		for(i=0; i<m*n; i++)
		{
			fscanf(fp, "%d", &V[i]);
		}
		// copy host values to devices copies
		hipMemcpy(d_V, V, m*n*sizeof(int), hipMemcpyHostToDevice);

		dim3 grid(m, m);
		// compute the execution time
		hipEvent_t start, stop;
		// create event
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// record event
		hipEventRecord(start);
		// launch knn() kernel on GPU
		computeDist<<<grid, n, n*sizeof(int)>>>(m, n, d_V, D);
		hipDeviceSynchronize();
		knn<<<m, m, m*sizeof(int)>>>(m, n, k, d_V, D, d_out);
		// record event and synchronize
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float time;
		// get event elapsed time
		hipEventElapsedTime(&time, start, stop);
//		fprintf(fp_out, "GPU calculation time:%f ms\n", time);
		// copy result back to host
		hipMemcpy(out, d_out, m*k*sizeof(int), hipMemcpyDeviceToHost);

		showResult(m, k, out);
		printf("%f\n", time);
		// cleanup
		hipFree(d_V);
		hipFree(d_out);
		hipFree(D);

		free(V);
		free(out);
	}
//	fclose(fp_in);
//	fclose(fp_out);
	fclose(fp);
	return 0;
}

