#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define INIT_MAX 100
void showResult(int m, int k, int *out);

/* 
* m: total num of points
* n: n dimensions
* k: num of nearest points
* V: point coordinates
* out: k nearest neighbors
*/

__global__ void knn(int m, int n, int k, int *V, int *out)
{
	int i,j;
	int dim;
	int temp;
	int sum;
	int count;
	int last_idx;
//	int m = (*d_m);
//	int n = (*d_n);
//	int k = (*d_k);
	for(i=0; i<m; i++)
	{
		temp = INIT_MAX;
		last_idx = i;
		for(count=0; count<k; count++)
		{
			for(j=0; j<m; j++)
			{
				sum = 0;
				if(j != last_idx && j != i)
				{
					for(dim=0; dim<n; dim++)
					{
						sum+=(V[i*n+dim]-V[j*n+dim])*(V[i*n+dim]-V[j*n+dim]);
					}
					if(sum < temp)
					{
						temp = sum;
						out[i*k+count] = j;
						last_idx = j;
					}
				}
			}
		}
	}
//	showResult(m, k, out);
}

//__host__ __device__ void showResult(int m, int k, int *out)
void showResult(int m, int k, int *out)
{
	int i,j;
	for(i=0; i<m; i++)
	{
		for(j=0; j<k; j++)
		{
				printf("%d", out[i*k+j]);
				if(j == k-1)
				{
						printf("\n");
				}
				else
				{
						printf(" ");
				}
		}
	} 
} 
int main(int argc, char *argv[]) 
{ 
	int m,n,k;
//	int *d_m, *d_n, *d_k;
	int i,j;
	int *V, *out;					//host copies
	int *d_V, *d_out;			//device copies
	FILE *fp;
	if(argc != 2)
	{
		printf("Usage: knn [file path]\n");
		exit(1);
	}
	if((fp = fopen(argv[1], "r")) == NULL)
	{
		printf("Error open file!\n");
		exit(1);
	}
	while(fscanf(fp, "%d %d %d", &m, &n, &k) != EOF)
	{
		printf("m:%d, n:%d, k:%d\n", m, n, k);

//		cudaMalloc((void **)&d_m, sizeof(int));
//		cudaMalloc((void **)&d_n, sizeof(int));
//		cudaMalloc((void **)&d_k, sizeof(int));
//
//		cudaMemcpy(d_m, &m, sizeof(int), cudaMemcpyHostToDevice);
//		cudaMemcpy(d_n, &n, sizeof(int), cudaMemcpyHostToDevice);
//		cudaMemcpy(d_k, &k, sizeof(int), cudaMemcpyHostToDevice);

		V = (int *) malloc(m*n*sizeof(int));
		out = (int *) malloc(m*k*sizeof(int));

		hipMalloc((void **)&d_V, m*n*sizeof(int));
		hipMalloc((void **)&d_out, m*k*sizeof(int));

		for(i=0; i<m*n; i++)
		{
			fscanf(fp, "%d", &V[i]);
		}
		
		hipMemcpy(d_V, V, m*n*sizeof(int), hipMemcpyHostToDevice);

		knn<<<1,1>>>(m, n, k, d_V, d_out);

		hipMemcpy(out, d_out, m*k*sizeof(int), hipMemcpyDeviceToHost);

		for(i=0; i<m; i++)
		{
				for(j=0; j<k; j++)
				{
						printf("%d", out[i*k+j]);
						if(j == k-1)
						{
								printf("\n");
						}
						else
						{
								printf(" ");
						}
				}
		}
//		cudaFree(d_m);
//		cudaFree(d_n);
//		cudaFree(d_k);
		hipFree(d_V);
		hipFree(d_out);

		free(V);
		free(out);

		V = NULL;
		out = NULL;
	}
	fclose(fp);
	return 0;
}

