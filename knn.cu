/* 
* INPUT:
* m: total num of points
* m is in [10, 1000]
* n: n dimensions
* n is in [1,1000]
* k: num of nearest points
* k is in [1,10]
* V: point coordinates
* the integer elements are in [-5,5]
* OUTPUT:
* out: k nearest neighbors
*/

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define INIT_MAX 100000
void showResult(int m, int k, int *out);

extern __shared__ int SM[];

// compute the square of distance per dimension
// the kth dimension of the ith point and jth point
__device__ void computeDimDist(int i, int j, int n, int *V)
{
	int k = threadIdx.x;
	SM[k] = (V[i*n+k]-V[j*n+k])*(V[i*n+k]-V[j*n+k]);
}

// compute the square of distance of the ith point and jth point
__global__ void computeDist(int m, int n, int *V, int *D)
{
	int i = blockIdx.x;
   	int j = blockIdx.y;
	int k = threadIdx.x;
	int s;
	// calculate the square of distance per dimensions
	// reduce duplicated calculations since d(i, j) = d(j, i)
	// also, we do not consider the trivial case of d(i, i) = 0
	// so we only compute the square distance when i < j 
	if(i < j)
	{
		computeDimDist(i, j, n, V);
		__syncthreads();
		// use paralel reduction
		for(s=n/2; s>0; s>>=1)
		{
			if(k < s)
			{
				SM[k] += SM[k+s];
			}
			__syncthreads();
		}
		if(k == 0)
		{
			// when n is odd, the last element of SM needs to be added
			if(n > (n/2)*2)
			{
				D[i*m+j] = SM[0] + SM[n-1];
			}
			else
			{
				D[i*m+j] = SM[0];
			}
		}
	}
}

__device__ void initSM(int m, int *D)
{
	int i = blockIdx.x;
	int j = threadIdx.x;
	if(i < j)
	{
		SM[j] = D[i*m+j];
	}
	else
	{
		SM[j] = D[j*m+i];	
	}
	__syncthreads();
}

// compute the k nearest neighbors
__global__ void knn(int m, int n, int k, int *V, int *D, int *out)
{
	int i,j;
	int temp;
	int count;
	int num;
	int dist;
	int is_duplicate;

	// find the k nearest neighbors of the point with index = blockIdx.x
	i = blockIdx.x;
	
	initSM(m, D);
	__syncthreads();

	// let the first thread select the k-min distance
	if(threadIdx.x == 0)
	{
		for(count=0; count<k; count++)
		{
			temp = INIT_MAX;
			// iterate the jth point
			for(j=0; j<m; j++)
			{
				// check whether the jth point is the same point as the ith one
				// or has already in the k-nn list
				is_duplicate = 0;
				if(j == i)
				{
					is_duplicate = 1;
				}
				for(num=0; num<count; num++)
				{
					if(out[i*k+num] == j)
					{
						is_duplicate = 1;
					}
				}
				if(!is_duplicate)
				{
					dist = SM[j];
					if(dist < temp)
					{
						temp = dist;
						out[i*k+count] = j;
					}
				}
			}
		}
	}
}

void showResult(int m, int k, int *out)
{
	int i,j;
	for(i=0; i<m; i++)
	{
		for(j=0; j<k; j++)
		{
			printf("%d ", out[i*k+j]);
			if(j == k-1)
			{
				printf("\n");
			}
			/*
			else
			{
				printf(" ");
			}
			*/
		}
	} 
} 
int main(int argc, char *argv[]) 
{ 
	int m,n,k;
	int i;
	int *V, *out;				//host copies
	int *d_V, *d_out;			//device copies
	int *D;						
	FILE *fp_in;
	FILE *fp_out;
	if(argc != 2)
	{
		printf("Usage: knn inputfile\n");
		exit(1);
	}
	if((fp_in = fopen(argv[1], "r")) == NULL)
	{
		printf("Error open input file!\n");
		exit(1);
	}
	if((fp_out = fopen("time.txt", "w")) == NULL)
	{
		printf("Error open output file!\n");
		exit(1);
	}
	while(fscanf(fp_in, "%d %d %d", &m, &n, &k) != EOF)
	{
		V = (int *) malloc(m*n*sizeof(int));
		out = (int *) malloc(m*k*sizeof(int));
		// allocate space for devices copies
		hipMalloc((void **)&d_V, m*n*sizeof(int));
		hipMalloc((void **)&d_out, m*k*sizeof(int));
		hipMalloc((void **)&D, m*m*sizeof(int));

		for(i=0; i<m*n; i++)
		{
			fscanf(fp_in, "%d", &V[i]);
		}
		// copy host values to devices copies
		hipMemcpy(d_V, V, m*n*sizeof(int), hipMemcpyHostToDevice);

		dim3 grid(m, m);
		// compute the execution time
		hipEvent_t start, stop;
		// create event
		hipEventCreate(&start);
		hipEventCreate(&stop);
		// record event
		hipEventRecord(start);
		// launch knn() kernel on GPU
		computeDist<<<grid, n, n*sizeof(int)>>>(m, n, d_V, D);
		hipDeviceSynchronize();
		knn<<<m, m, m*sizeof(int)>>>(m, n, k, d_V, D, d_out);
		// record event and synchronize
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		float time;
		// get event elapsed time
		hipEventElapsedTime(&time, start, stop);
		fprintf(fp_out, "GPU calculation time:%f ms\n", time);
		// copy result back to host
		hipMemcpy(out, d_out, m*k*sizeof(int), hipMemcpyDeviceToHost);

		showResult(m, k, out);
		// cleanup
		hipFree(d_V);
		hipFree(d_out);
		hipFree(D);

		free(V);
		free(out);
	}
	fclose(fp_in);
	fclose(fp_out);
	return 0;
}

