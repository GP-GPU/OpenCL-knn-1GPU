/* 
* m: total num of points
* n: n dimensions
* k: num of nearest points
* V: point coordinates
* out: k nearest neighbors
*/

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<stdlib.h>

#define INIT_MAX 100
void showResult(int m, int k, int *out);

extern __shared__ int D[];

__device__ void computeDist(int m, int n, int *V)
{
	int i=threadIdx.x;
   	int j=threadIdx.y;
	int k;
	int dist = 0;
	//reduce duplications
	if(i < j)
	{
		for(k=0; k<n; k++)
		{
			dist += (V[i*n+k]-V[j*n+k])*(V[i*n+k]-V[j*n+k]);
		}
	}
	D[i*m+j] = dist;
}

//paralle reduction?
//cannot use D now!
__device__ int prmin(int m, int *D, int *R)
{
	int j = threadIdx.y;
	int s = blockDim.y/2;
	R[j] = j;
	__syncthreads();
	for(s=blockDim.y; s>0; s>>=1)
	{
		if(j < s)
		{
			D[j] = D[j]<D[j+s]? D[j]: D[j+s]; 
			R[j] = R[j]<R[j+s]? R[j]: R[j+s]; 
		}
		__syncthreads();
	}
	return R[0];
}

__global__ void knn(int m, int n, int k, int *V, int *out)
{
	int i,j;
	int dim;
	int temp;
	int sum;
	int count;
	int num;
	int dist;
	int is_duplicate;
//	__shared__ int D[m*m];

	computeDist(m, n, V);
	__syncthreads();
	// let the first thread select the k-min dist
	i = threadIdx.x;
	if(threadIdx.y == 0)
	{
		for(count=0; count<k; count++)
		{
			temp = INIT_MAX;
			for(j=0; j<m; j++)
			{
				is_duplicate = 0;

				if(j == i)
				{
					is_duplicate = 1;
				}
				
				for(num=0; num<count; num++)
				{
					if(out[i*k+num] == j)
					{
						is_duplicate = 1;
					}
				}
	
				if(!is_duplicate)
				{
					if(i < j)
					{
						dist = D[i*m+j];
					}
					else
					{
						dist = D[j*m+i];
					}
					if(dist < temp)
					{
						temp = dist;
						out[i*k+count] = j;
					}
				}
			}
		}
	}
}

void showResult(int m, int k, int *out)
{
	int i,j;
	for(i=0; i<m; i++)
	{
		for(j=0; j<k; j++)
		{
			printf("%d", out[i*k+j]);
			if(j == k-1)
			{
				printf("\n");
			}
			else
			{
				printf(" ");
			}
		}
	} 
} 
int main(int argc, char *argv[]) 
{ 
	int m,n,k;
	int i;
	int *V, *out;				//host copies
	int *d_V, *d_out;			//device copies
	
//	int *D;						//will be replaced with shared memory

	FILE *fp;
	if(argc != 2)
	{
		printf("Usage: knn [file path]\n");
		exit(1);
	}
	if((fp = fopen(argv[1], "r")) == NULL)
	{
		printf("Error open file!\n");
		exit(1);
	}
	while(fscanf(fp, "%d %d %d", &m, &n, &k) != EOF)
	{
		printf("m:%d, n:%d, k:%d\n", m, n, k);

		V = (int *) malloc(m*n*sizeof(int));
		out = (int *) malloc(m*k*sizeof(int));

		hipMalloc((void **)&d_V, m*n*sizeof(int));
		hipMalloc((void **)&d_out, m*k*sizeof(int));

//		cudaMalloc((void **)&D, m*m*sizeof(int));

		for(i=0; i<m*n; i++)
		{
			fscanf(fp, "%d", &V[i]);
		}
		
		hipMemcpy(d_V, V, m*n*sizeof(int), hipMemcpyHostToDevice);

		dim3 blk(m, m);
		knn<<<1, blk, m*m*sizeof(int)>>>(m, n, k, d_V, d_out);

		hipMemcpy(out, d_out, m*k*sizeof(int), hipMemcpyDeviceToHost);

		showResult(m, k, out);

		hipFree(d_V);
		hipFree(d_out);

		free(V);
		free(out);
	}
	fclose(fp);
	return 0;
}

